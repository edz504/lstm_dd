
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

/////////////////// by chenyi
extern vector<float> output_blob;
/////////////////// by chenyi

template <typename Dtype>
void CuDNNSigmoidHistLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        CUDNN_ACTIVATION_SIGMOID,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));

/////////////////// by chenyi
  const int count = bottom[0]->count();
  const Dtype* top_blob = top[0]->cpu_data();
  for (int i=0;i<count;i++) {
      output_blob[i]=top_blob[i];
  }
/////////////////// by chenyi
}

template <typename Dtype>
void CuDNNSigmoidHistLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        CUDNN_ACTIVATION_SIGMOID,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSigmoidHistLayer);

}  // namespace caffe
#endif
